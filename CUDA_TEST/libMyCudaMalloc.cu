#include <stdio.h>
#include <hip/hip_runtime.h>
#include "libMyCudaMalloc.h"

hipError_t myCudaMalloc(void** devPtr, size_t size){
	return hipMalloc(devPtr, size);
}

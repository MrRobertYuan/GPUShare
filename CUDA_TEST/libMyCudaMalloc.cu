#include <stdio.h>
#include <hip/hip_runtime.h>
#include "libMyCudaMalloc.h"

hipError_t myCudaMalloc(void** devPtr, size_t size){
	return hipMalloc(devPtr, size);
}

hipError_t myCudaFree(void* devPtr){
	return hipFree(devPtr);
}

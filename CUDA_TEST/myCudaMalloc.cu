#include <stdio.h>
#include <hip/hip_runtime.h>
#include "myCudaMalloc.h"

hipError_t myCudaMalloc(void** devPtr, size_t size){
	return hipMalloc(devPtr, size);
}

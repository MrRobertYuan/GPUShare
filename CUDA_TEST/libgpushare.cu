#include <stdio.h>
#include <hip/hip_runtime.h>
#include "libgpushare.h"
#include "libMyCudaMalloc.h"

const size_t maxSize = 20000000;
size_t allocatedSize = 0;

const int mod = 9999987;
long long keys[mod + 100];
size_t values[mod + 100];

void Insert(long long key, size_t value){
	int hashKey = key % mod;
	while(keys[hashKey] != 0 && keys[hashKey] != key){
		hashKey ++;
	}
	keys[hashKey] = key;
	values[hashKey] = value;	
}

size_t Query(long long key){
	int hashKey = key % mod;
	while(keys[hashKey] != 0 && keys[hashKey] != key){
		hashKey ++;
	}
	if(keys[hashKey] == key){
		return values[hashKey];
	}
	return 0;
}

hipError_t hipMalloc(void **devPtr, size_t size)
{
	if(allocatedSize + size <= maxSize){
		hipError_t res = myCudaMalloc(devPtr, size);
		if(res == hipSuccess){
			allocatedSize += size;
			Insert((long long)*devPtr, size);
		}
		return res;
	}
	else{
		*devPtr = NULL;
		return hipErrorOutOfMemory;
	}
}

hipError_t hipFree(void *devPtr){
	hipError_t res = myCudaFree(devPtr);	
	if(res == hipSuccess){
		allocatedSize -= Query((long long)devPtr);
	}
	return res;
}

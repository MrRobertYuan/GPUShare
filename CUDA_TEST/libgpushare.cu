#include <stdio.h>
#include <hip/hip_runtime.h>
#include "libgpushare.h"
#include "libMyCudaMalloc.h"

size_t maxSize = 20000000;

hipError_t hipMalloc(void **devPtr, size_t size)
{
	if(size <= maxSize){
		return myCudaMalloc(devPtr, size);
	}
	else{
		*devPtr = NULL;
		return hipErrorOutOfMemory;
	}
}
